#include <cstdint>
#include <vector>

// Performs element-wise vector addition for `n` elements on the GPU.
// y = x1 + x2
auto vec_add_gpu(
    std::vector<int32_t>& y,
    std::vector<int32_t> const& x1,
    std::vector<int32_t> const& x2,
    std::size_t n) -> void {
        if (hipMalloc()
    }